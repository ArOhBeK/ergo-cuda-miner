#include "hip/hip_runtime.h"
#include "autolykos2_cuda_miner.h"
#include "blake2b_cuda.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <string.h>
#include <stdint.h>

// Autolykos2 constants
#define AUTOLYKOS2_N 26  // Memory parameter (2^26 elements)
#define AUTOLYKOS2_K 32  // Hash length
#define AUTOLYKOS2_M (1 << AUTOLYKOS2_N) // 67,108,864 elements
#define BLOCK_SIZE 256
#define GRID_SIZE 1024

// Constants from the cloned repository (adjust if needed)
#define NONCES_PER_ITER (BLOCK_SIZE * GRID_SIZE) // Assuming each thread processes one nonce per launch
#define THREADS_PER_ITER BLOCK_SIZE // Assuming block size is threads per iteration
#define NUM_SIZE_32 8 // Corresponds to 32 bytes (Blake2b hash size)
#define K_LEN 64 // Corresponds to 64 indices in Autolykos2
#define BUF_SIZE_8 128 // Buffer size in bytes (Blake2b block size)
#define MAX_SOLS 10 // Maximum number of solutions to find

// B2B_IV macro from the cloned repository
#define B2B_IV(h) \
    do { \
        ((uint64_t *)(h))[0] = 0x6A09E667F2BDC928ULL; \
        ((uint64_t *)(h))[1] = 0xBB67AE8584CAA73BULL; \
        ((uint64_t *)(h))[2] = 0x3C6EF372FE94F82BULL; \
        ((uint64_t *)(h))[3] = 0xA54FF53A5F1D36F1ULL; \
        ((uint64_t *)(h))[4] = 0x510E527FADE682D1ULL; \
        ((uint64_t *)(h))[5] = 0x9B05688C2B3E6C1FULL; \
        ((uint64_t *)(h))[6] = 0x1F83D9ABFB41BD6BULL; \
        ((uint64_t *)(h))[7] = 0x5BE0CD19137E2179ULL; \
    } while(0)

// Helper device functions from the cloned repository
__device__ __forceinline__ uint32_t ld_gbl_cs(const  uint32_t * __restrict__ p) {
	uint32_t v;
	asm("ld.global.cs.u32 %0, [%1];" : "=r"(v) : "l"(p));
	return v;
}

__device__ __forceinline__ uint4 ld_gbl_cs_v4(const  uint4 * __restrict__ p) {
	uint4 v;
	asm("ld.global.cs.v4.u32 {%0, %1, %2, %3}, [%4];" : "=r"(v.x), "=r"(v.y), "=r"(v.z), "=r"(v.w) : "l"(p));
	return v;
}

__device__ __forceinline__ uint32_t cuda_swab32(uint32_t x)
{
	/* device */
	return __byte_perm(x, x, 0x0123);
}

__device__ __forceinline__ uint64_t devectorize(uint2 x)
{
	uint64_t result;
	asm("mov.b64 %0,{%1,%2}; \n\t"
		: "=l"(result) : "r"(x.x), "r"(x.y));
	return result;
}


__device__ __forceinline__ uint2 vectorize(const uint64_t x)
{
	uint2 result;
	asm("mov.b64 {%0,%1},%2; \n\t"
		: "=r"(result.x), "=r"(result.y) : "l"(x));
	return result;
}

__device__ __forceinline__
uint64_t devROTR64(uint64_t b, int offset)
{
	uint2 a;
	uint2 result;
	a = vectorize(b);

	if (offset < 32) {
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
	}
	else {
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
	}
	return devectorize(result);
}

__device__ __forceinline__
void devB2B_G(uint64_t* v, int a, int b, int c, int d, uint64_t x, uint64_t y)
{
    ((uint64_t *)(v))[a] += ((uint64_t *)(v))[b] + x;
    ((uint64_t *)(v))[d]
        = devROTR64(((uint64_t *)(v))[d] ^ ((uint64_t *)(v))[a], 32);
    ((uint64_t *)(v))[c] += ((uint64_t *)(v))[d];
    ((uint64_t *)(v))[b]
        = devROTR64(((uint64_t *)(v))[b] ^ ((uint64_t *)(v))[c], 24);
    ((uint64_t *)(v))[a] += ((uint64_t *)(v))[b] + y;
    ((uint64_t *)(v))[d]
        = devROTR64(((uint64_t *)(v))[d] ^ ((uint64_t *)(v))[a], 16);
    ((uint64_t *)(v))[c] += ((uint64_t *)(v))[d];
    ((uint64_t *)(v))[b]
        = devROTR64(((uint64_t *)(v))[b] ^ ((uint64_t *)(v))[c], 63);
}


__device__ __forceinline__
void devB2B_MIX(uint64_t* v, uint64_t* m)
{
	devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[0], ((uint64_t *)(m))[1]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[2], ((uint64_t *)(m))[3]);
	devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[4], ((uint64_t *)(m))[5]);
	devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[6], ((uint64_t *)(m))[7]);
	devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[8], ((uint64_t *)(m))[9]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[10], ((uint64_t *)(m))[11]);
    devB2B_G(v, 2, 7,  8, 13, ((uint64_t *)(m))[12], ((uint64_t *)(m))[13]);
    devB2B_G(v, 3, 4,  9, 14, ((uint64_t *)(m))[14], ((uint64_t *)(m))[15]);

    devB2B_G(v, 0, 4,  8, 12, ((uint64_t *)(m))[14], ((uint64_t *)(m))[10]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[4], ((uint64_t *)(m))[8]);
	devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[9], ((uint64_t *)(m))[15]);
	devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[13], ((uint64_t *)(m))[6]);
	devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[1], ((uint64_t *)(m))[12]);
	devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[0], ((uint64_t *)(m))[2]);
	devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[11], ((uint64_t *)(m))[7]);
	devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[5], ((uint64_t *)(m))[3]);

	devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[11], ((uint64_t *)(m))[8]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[12], ((uint64_t *)(m))[0]);
	devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[5], ((uint64_t *)(m))[2]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[15], ((uint64_t *)(m))[13]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[10], ((uint64_t *)(m))[14]);
	devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[3], ((uint64_t *)(m))[6]);
	devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[7], ((uint64_t *)(m))[1]);
	devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[9], ((uint64_t *)(m))[4]);

	devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[7], ((uint64_t *)(m))[9]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[3], ((uint64_t *)(m))[1]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[13], ((uint64_t *)(m))[12]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[11], ((uint64_t *)(m))[14]);
	devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[2], ((uint64_t *)(m))[6]);
	devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[5], ((uint64_t *)(m))[10]);
	devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[4], ((uint64_t *)(m))[0]);
	devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[15], ((uint64_t *)(m))[8]);

	devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[9], ((uint64_t *)(m))[0]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[5], ((uint64_t *)(m))[7]);
	devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[2], ((uint64_t *)(m))[4]);
    devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[10], ((uint64_t *)(m))[15]);
	devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[14], ((uint64_t *)(m))[1]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[11], ((uint64_t *)(m))[12]);
	devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[6], ((uint64_t *)(m))[8]);
	devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[3], ((uint64_t *)(m))[13]);

	devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[2], ((uint64_t *)(m))[12]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[6], ((uint64_t *)(m))[10]);
	devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[0], ((uint64_t *)(m))[11]);
	devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[8], ((uint64_t *)(m))[3]);
	devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[4], ((uint64_t *)(m))[13]);
	devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[7], ((uint64_t *)(m))[5]);
    devB2B_G(v, 2, 7,  8, 13, ((uint64_t *)(m))[15], ((uint64_t *)(m))[14]);
	devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[1], ((uint64_t *)(m))[9]);

	devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[12], ((uint64_t *)(m))[5]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[1], ((uint64_t *)(m))[15]);
    devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[14], ((uint64_t *)(m))[13]);
	devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[4], ((uint64_t *)(m))[10]);
	devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[0], ((uint64_t *)(m))[7]);
	devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[6], ((uint64_t *)(m))[3]);
	devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[9], ((uint64_t *)(m))[2]);
	devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[8], ((uint64_t *)(m))[11]);

    devB2B_G(v, 0, 4,  8, 12, ((uint64_t *)(m))[13], ((uint64_t *)(m))[11]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[7], ((uint64_t *)(m))[14]);
	devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[12], ((uint64_t *)(m))[1]);
	devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[3], ((uint64_t *)(m))[9]);
	devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[5], ((uint64_t *)(m))[0]);
	devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[15], ((uint64_t *)(m))[4]);
	devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[8], ((uint64_t *)(m))[6]);
	devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[2], ((uint64_t *)(m))[10]);

	devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[6], ((uint64_t *)(m))[15]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[14], ((uint64_t *)(m))[9]);
	devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[11], ((uint64_t *)(m))[3]);
	devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[0], ((uint64_t *)(m))[8]);
	devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[12], ((uint64_t *)(m))[2]);
	devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[13], ((uint64_t *)(m))[7]);
	devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[1], ((uint64_t *)(m))[4]);
	devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[10], ((uint64_t *)(m))[5]);

	devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[10], ((uint64_t *)(m))[2]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[8], ((uint64_t *)(m))[4]);
	devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[7], ((uint64_t *)(m))[6]);
	devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[1], ((uint64_t *)(m))[5]);
    devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[15], ((uint64_t *)(m))[11]);
	devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[9], ((uint64_t *)(m))[14]);
	devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[3], ((uint64_t *)(m))[12]);
	devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[13], ((uint64_t *)(m))[0]);

	devB2B_G(v, 0, 4, 8, 12, ((uint64_t *)(m))[0], ((uint64_t *)(m))[1]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[2], ((uint64_t *)(m))[3]);
	devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[4], ((uint64_t *)(m))[5]);
	devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[6], ((uint64_t *)(m))[7]);
	devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[8], ((uint64_t *)(m))[9]);
    devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[10], ((uint64_t *)(m))[11]);
    devB2B_G(v, 2, 7,  8, 13, ((uint64_t *)(m))[12], ((uint64_t *)(m))[13]);
    devB2B_G(v, 3, 4,  9, 14, ((uint64_t *)(m))[14], ((uint64_t *)(m))[15]);

    devB2B_G(v, 0, 4,  8, 12, ((uint64_t *)(m))[14], ((uint64_t *)(m))[10]);
	devB2B_G(v, 1, 5, 9, 13, ((uint64_t *)(m))[4], ((uint64_t *)(m))[8]);
	devB2B_G(v, 2, 6, 10, 14, ((uint64_t *)(m))[9], ((uint64_t *)(m))[15]);
	devB2B_G(v, 3, 7, 11, 15, ((uint64_t *)(m))[13], ((uint64_t *)(m))[6]);
	devB2B_G(v, 0, 5, 10, 15, ((uint64_t *)(m))[1], ((uint64_t *)(m))[12]);
	devB2B_G(v, 1, 6, 11, 12, ((uint64_t *)(m))[0], ((uint64_t *)(m))[2]);
	devB2B_G(v, 2, 7, 8, 13, ((uint64_t *)(m))[11], ((uint64_t *)(m))[7]);
	devB2B_G(v, 3, 4, 9, 14, ((uint64_t *)(m))[5], ((uint64_t *)(m))[3]);
}

// Constant memory for Blake2b IVs
const __constant__ uint64_t ivals[8] = {
    0x6A09E667F2BDC928,
    0xBB67AE8584CAA73B,
    0x3C6EF372FE94F82B,
    0xA54FF53A5F1D36F1,
    0x510E527FADE682D1,
    0x9B05688C2B3E6C1F,
    0x1F83D9ABFB41BD6B,
    0x5BE0CD19137E2179
};

// Constant memory for the target boundary
__constant__ uint8_t bound_[32]; // Assuming target is 32 bytes

// Function to copy the target boundary to constant memory
void cpyBSymbol(uint8_t *bound)
{
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(bound_), bound, NUM_SIZE_32 * sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in cpyBSymbol: %s\n", hipGetErrorString(err));
    }
}


// Autolykos2 mining kernel (combined logic from BlockMiningStep1 and BlockMiningStep2)
__global__ void autolykos2_mining_kernel(
    const uint32_t* dataset, // Corresponds to 'hashes' in the cloned repo
    const uint8_t* header,   // Corresponds to 'data' in the cloned repo (first 76 bytes)
    uint64_t start_nonce,    // Corresponds to 'base' in the cloned repo
    uint32_t target_hi,      // Not directly used in the new kernel logic, target is in bound_
    uint32_t* d_found_nonce_param,
    bool* d_found_flag_param
) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

	uint64_t aux[32] = { 0 };
	uint32_t ind[K_LEN] = { 0 }; // Use K_LEN for index array size
	uint32_t r[NUM_SIZE_32 + 1] = { 0 }; // Use NUM_SIZE_32 for hash size + 1 for carry

    uint8_t j = 0;

    if (tid < NONCES_PER_ITER) // Check if thread ID is within the total number of nonces to process
    {
        uint64_t nonce = start_nonce + tid; // Calculate the specific nonce for this thread

        // Prepare mining input: header (76 bytes) + nonce (8 bytes)
        uint8_t mining_input[84];

        // Copy header
        for (int i = 0; i < 76; i++) {
            mining_input[i] = header[i];
        }

        // Add nonce (little-endian)
        mining_input[76] = nonce & 0xFF;
        mining_input[77] = (nonce >> 8) & 0xFF;
        mining_input[78] = (nonce >> 16) & 0xFF;
        mining_input[79] = (nonce >> 24) & 0xFF;
        mining_input[80] = (nonce >> 32) & 0xFF;
        mining_input[81] = (nonce >> 40) & 0xFF;
        mining_input[82] = (nonce >> 48) & 0xFF;
        mining_input[83] = (nonce >> 56) & 0xFF;

        // First Blake2b hash (using the existing blake2b_cuda function)
        uint8_t hash1[32];
        blake2b_cuda(hash1, mining_input, 84);

        // --- Start of BlockMiningStep1 logic ---

        uint64_t tmp;
        ((uint32_t*)(&tmp))[0] = cuda_swab32(((uint32_t*)&nonce)[1]); // Swap bytes for little-endian
        ((uint32_t*)(&tmp))[1] = cuda_swab32(((uint32_t*)&nonce)[0]); // Swap bytes for little-endian

        B2B_IV(aux);
        B2B_IV(aux + 8);
        aux[0] = ivals[0];
        ((uint64_t *)(aux))[12] ^= 40; // Message length (76 header + 8 nonce = 84 bytes, Blake2b processes in 128-byte blocks, so 84 is padded to 128. The length field in Blake2b is the total number of bytes processed. The original code uses 40, which might be related to an internal state or a different message length calculation. Let's stick to 40 for now as in the original code.)
        ((uint64_t *)(aux))[13] ^= 0;

        ((uint64_t *)(aux))[14] = ~((uint64_t *)(aux))[14];

        ((uint64_t *)(aux))[16] = ((uint64_t *)hash1)[0]; // Copy hash1 (32 bytes)
        ((uint64_t *)(aux))[17] = ((uint64_t *)hash1)[1];
        ((uint64_t *)(aux))[18] = ((uint64_t *)hash1)[2];
        ((uint64_t *)(aux))[19] = ((uint64_t *)hash1)[3];
        ((uint64_t *)(aux))[20] = tmp; // Add nonce (swapped)
        ((uint64_t *)(aux))[21] = 0;
        ((uint64_t *)(aux))[22] = 0;
        ((uint64_t *)(aux))[23] = 0;
        ((uint64_t *)(aux))[24] = 0;
        ((uint64_t *)(aux))[25] = 0;
        ((uint64_t *)(aux))[26] = 0;
        ((uint64_t *)(aux))[27] = 0;
        ((uint64_t *)(aux))[28] = 0;
        ((uint64_t *)(aux))[29] = 0;
        ((uint64_t *)(aux))[30] = 0;
        ((uint64_t *)(aux))[31] = 0;

        devB2B_MIX(aux, aux + 16);

        uint64_t hsh;
#pragma unroll
        for (j = 0; j < NUM_SIZE_32; j += 2)
        {
            hsh = ivals[j >> 1];
            hsh ^= ((uint64_t *)(aux))[j >> 1] ^ ((uint64_t *)(aux))[8 + (j >> 1)];

            ((uint32_t*)r)[j] =  ((uint32_t*)(&hsh))[0];
            ((uint32_t*)r)[j + 1] = ((uint32_t*)(&hsh))[1];
        }

        // --- End of BlockMiningStep1 logic ---
        // --- Start of BlockMiningStep2 logic ---

        // Generate indices from hash result
        uint32_t n_len = AUTOLYKOS2_M; // Dataset size
        for (int k = 0; k < K_LEN; k++) {
             uint32_t val;
             int byte_idx = (k / 4) * 4; // Index into r_bytes (0, 4, 8, ..., 28)
             if (byte_idx + 3 < 32) { // Ensure we don't go out of bounds
                 val = ((uint32_t*)r)[byte_idx/4];
             } else {
                 byte_idx = byte_idx % 32;
                 val = ((uint32_t*)r)[byte_idx/4];
             }

             // Apply shifts based on k % 4
             int sub_idx = k % 4;
             uint32_t final_val;
             if (sub_idx == 0) final_val = val;
             else if (sub_idx == 1) final_val = (val << 8) | (val >> 24);
             else if (sub_idx == 2) final_val = (val << 16) | (val >> 16);
             else final_val = (val << 24) | (val >> 8);

             ind[k] = final_val % n_len;
        }

        // Calculate result (summation from dataset)
        uint32_t current_sum[NUM_SIZE_32 + 1] = {0}; // Sum stored as uint32_t array for carry

        // Add dataset elements based on indices
        for (int k = 0; k < K_LEN; ++k)
        {
            uint32_t dataset_element = dataset[ind[k]];

            // Add dataset_element to current_sum (uint32_t array addition with carry)
            uint64_t temp_sum = (uint64_t)current_sum[0] + dataset_element;
            current_sum[0] = (uint32_t)temp_sum;
            uint32_t carry = temp_sum >> 32;

            for (int i = 1; i < NUM_SIZE_32 + 1; ++i) {
                temp_sum = (uint64_t)current_sum[i] + carry;
                current_sum[i] = (uint32_t)temp_sum;
                carry = temp_sum >> 32;
            }
        }

        // Convert the uint32_t sum array back to a 32-byte hash format for the final Blake2b
        uint8_t sum_bytes[32];
        for(int i = 0; i < NUM_SIZE_32; ++i) {
            sum_bytes[i*4] = current_sum[i] & 0xFF;
            sum_bytes[i*4 + 1] = (current_sum[i] >> 8) & 0xFF;
            sum_bytes[i*4 + 2] = (current_sum[i] >> 16) & 0xFF;
            sum_bytes[i*4 + 3] = (current_sum[i] >> 24) & 0xFF;
        }

        // Final hash with sum
        uint8_t final_input[40]; // 32 bytes hash1 + 8 bytes sum
        for (int i = 0; i < 32; i++) {
            final_input[i] = hash1[i];
        }

        // Add sum (little-endian, 8 bytes) - using the first 8 bytes of sum_bytes
        for (int i = 0; i < 8; i++) {
            final_input[32 + i] = sum_bytes[i];
        }

        uint8_t final_hash[32];
        blake2b_cuda(final_hash, final_input, 40);

        // Check if hash meets target (compare with bound_)
        bool is_less_than_target = false;
        bool is_equal_to_target = true;

        for (int i = 31; i >= 0; --i) { // Compare from most significant byte
            if (final_hash[i] < bound_[i]) {
                is_less_than_target = true;
                is_equal_to_target = false;
                break;
            }
            if (final_hash[i] > bound_[i]) {
                is_equal_to_target = false;
                break;
            }
        }

        if (is_less_than_target || is_equal_to_target) {
            // Found valid nonce
            if (atomicCAS((int*)d_found_flag_param, 0, 1) == 0) {
                *d_found_nonce_param = (uint32_t)nonce; // Store the nonce
            }
        }
        // --- End of BlockMiningStep2 logic ---
    }
}

// CUDA error checking macro
#define CUDA_CHECK_INIT(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            return false; \
        } \
    } while(0)

// Device memory pointers
static uint32_t* d_dataset = nullptr;
static uint8_t* d_header = nullptr;
static uint32_t* d_found_nonce = nullptr;
static bool* d_found_flag = nullptr;
static uint8_t* d_target_boundary = nullptr;

// Host memory
static uint32_t* h_dataset = nullptr;
static bool miner_initialized = false;

// Generate Autolykos2 dataset on GPU
__global__ void generate_dataset_kernel(uint32_t* dataset, const uint8_t* seed, uint32_t start_idx, uint32_t count) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;

    uint32_t global_idx = start_idx + idx;

    // Generate dataset element using Blake2b
    uint8_t input[36]; // 32 bytes seed + 4 bytes index
    for (int i = 0; i < 32; i++) {
        input[i] = seed[i];
    }

    // Little-endian encoding of index
    input[32] = global_idx & 0xFF;
    input[33] = (global_idx >> 8) & 0xFF;
    input[34] = (global_idx >> 16) & 0xFF;
    input[35] = (global_idx >> 24) & 0xFF;

    uint8_t hash[32];
    blake2b_cuda(hash, input, 36);

    // Convert hash to uint32_t (little-endian)
    dataset[global_idx] =
        ((uint32_t)hash[0]) |
        ((uint32_t)hash[1] << 8) |
        ((uint32_t)hash[2] << 16) |
        ((uint32_t)hash[3] << 24);
}

bool autolykos2_cuda_init(int device_id) {
    if (miner_initialized) {
        return true;
    }

    // Set CUDA device
    CUDA_CHECK_INIT(hipSetDevice(device_id));

    // Allocate dataset memory on GPU (256MB)
    size_t dataset_size = AUTOLYKOS2_M * sizeof(uint32_t);
    CUDA_CHECK_INIT(hipMalloc(&d_dataset, dataset_size));

    // Allocate other GPU memory
    CUDA_CHECK_INIT(hipMalloc(&d_header, 76)); // Only need 76 bytes for header
    CUDA_CHECK_INIT(hipMalloc(&d_found_nonce, sizeof(uint32_t)));
    CUDA_CHECK_INIT(hipMalloc(&d_found_flag, sizeof(bool)));
    CUDA_CHECK_INIT(hipMalloc(&d_target_boundary, 32)); // Allocate memory for target boundary

    // Allocate host memory for dataset (for debugging/verification)
    h_dataset = (uint32_t*)malloc(dataset_size);
    if (!h_dataset) {
        fprintf(stderr, "Failed to allocate host dataset memory\n");
        return false;
    }

    miner_initialized = true;
    return true;
}

bool autolykos2_cuda_generate_dataset(const uint8_t* seed) {
    if (!miner_initialized) {
        fprintf(stderr, "Miner not initialized\n");
        return false;
    }

    // Allocate temporary seed memory on GPU
    uint8_t* d_temp_seed = nullptr;
    CUDA_CHECK_INIT(hipMalloc(&d_temp_seed, 32));

    // Copy seed to GPU
    CUDA_CHECK_INIT(hipMemcpy(d_temp_seed, seed, 32, hipMemcpyHostToDevice));

    // Generate dataset in chunks to avoid timeout
    const uint32_t chunk_size = 1024 * 1024; // 1M elements at a time
    const uint32_t total_elements = AUTOLYKOS2_M;

    for (uint32_t start = 0; start < total_elements; start += chunk_size) {
        uint32_t count = (chunk_size < total_elements - start) ? chunk_size : (total_elements - start);

        dim3 block(BLOCK_SIZE);
        dim3 grid((count + BLOCK_SIZE - 1) / BLOCK_SIZE);

        generate_dataset_kernel<<<grid, block>>>(d_dataset, d_temp_seed, start, count);
        CUDA_CHECK_INIT(hipGetLastError());
        CUDA_CHECK_INIT(hipDeviceSynchronize());

        // Progress indication
        if (start % (chunk_size * 10) == 0) {
            printf("Dataset generation: %.2f%%\n",
                   100.0f * (start + count) / total_elements);
        }
    }

    CUDA_CHECK_INIT(hipFree(d_temp_seed));

    printf("Dataset generation completed\n");
    return true;
}

bool autolykos2_cuda_mine(
    const uint8_t* header,
    uint64_t start_nonce,
    uint32_t nonce_count,
    uint32_t target_hi,
    const uint8_t* target_boundary,
    uint32_t* found_nonce,
    bool* found
) {
    if (!miner_initialized) {
        fprintf(stderr, "Miner not initialized\n");
        return false;
    }

    // Copy header to GPU
    CUDA_CHECK_INIT(hipMemcpy(d_header, header, 76, hipMemcpyHostToDevice));

    // Copy target boundary to constant memory
    cpyBSymbol((uint8_t*)target_boundary);

    // Reset found flag
    bool host_found = false;
    CUDA_CHECK_INIT(hipMemcpy(d_found_flag, &host_found, sizeof(bool), hipMemcpyHostToDevice));

    // Launch mining kernel
    dim3 block(BLOCK_SIZE);
    dim3 grid((nonce_count + BLOCK_SIZE - 1) / BLOCK_SIZE);

    autolykos2_mining_kernel<<<grid, block>>>(
        d_dataset,
        d_header,
        start_nonce,
        target_hi,
        d_found_nonce,
        d_found_flag
    );

    CUDA_CHECK_INIT(hipGetLastError());
    CUDA_CHECK_INIT(hipDeviceSynchronize());

    // Check if solution found
    CUDA_CHECK_INIT(hipMemcpy(&host_found, d_found_flag, sizeof(bool), hipMemcpyDeviceToHost));
    *found = host_found;

    if (host_found) {
        CUDA_CHECK_INIT(hipMemcpy(found_nonce, d_found_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost));
    }

    return true;
}

void autolykos2_cuda_cleanup() {
    if (!miner_initialized) {
        return;
    }

    // Free GPU memory
    if (d_dataset) hipFree(d_dataset);
    if (d_header) hipFree(d_header);
    if (d_found_nonce) hipFree(d_found_nonce);
    if (d_found_flag) hipFree(d_found_flag);
    if (d_target_boundary) hipFree(d_target_boundary);

    // Free host memory
    if (h_dataset) {
        free(h_dataset);
        h_dataset = nullptr;
    }

    // Reset pointers
    d_dataset = nullptr;
    d_header = nullptr;
    d_found_nonce = nullptr;
    d_found_flag = nullptr;
    d_target_boundary = nullptr;

    miner_initialized = false;
}

uint64_t autolykos2_cuda_get_hashrate() {
    // Return estimated hashrate based on GPU performance
    return GRID_SIZE * BLOCK_SIZE * 1000; // Rough estimate
}

bool autolykos2_cuda_is_initialized() {
    return miner_initialized;
}

// Bridge function for main.cpp compatibility
bool launchMiningKernel(
    const uint8_t* header,
    const uint8_t* target,
    uint64_t nonceStart,
    uint64_t nonceRange,
    uint64_t& foundNonce,
    uint8_t* foundHash
) {
    if (!miner_initialized) {
        fprintf(stderr, "Miner not initialized\n");
        return false;
    }

    uint32_t found_nonce_32;
    bool found = false;

    bool success = autolykos2_cuda_mine(
        header,
        nonceStart,
        (uint32_t)nonceRange,
        0, // target_hi not used
        target,
        &found_nonce_32,
        &found
    );

    if (success && found) {
        foundNonce = found_nonce_32;
        // foundHash would need to be computed if needed
        return true;
    }

    return false;
}
